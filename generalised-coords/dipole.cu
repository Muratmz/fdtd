/* Program to simulate propagation of a gaussian wave in dipole coordinates */


#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <fstream>

#define Sc_nu 0.2f
#define Sc_mu 0.2f
#define imp0 377.0f
#define EPSILON_R 1
#define MU_R 1

// These are currently fixed, but they should be user-modifiable
const int SIZE_NU = 128;
const int SIZE_MU = 128;
const int MAXTIME = 1000;
const int SOURCE_LOCATION = (SIZE_NU/2) * SIZE_MU + (SIZE_MU/2);
const int RI = 1;

// Define structure to be passed to kernel
struct Data {
    float *r;
    float *sin_theta;
    float *h_nu;
    float *h_phi;
    float *h_mu;
    float *E_nu;
    float *E_phi;
    float *H_nu;
    float *H_phi;
    float *H_mu;
    struct Data *d;
};

// Compute r, theta from nu, mu
__global__ void
compute_r_theta(float *r_matrix, float *sin_theta_matrix)
{
    // Get thread position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    if(x < SIZE_NU && y < SIZE_MU) {
        int nu = x;
        int mu = y;
        
        // Note: nu and mu are just x and y. This obviates the need for the
        // nu_matrix and mu_matrix
        
        // Find position to store r and theta
        float *r = r_matrix + x*SIZE_MU + y;
        float *sin_theta = sin_theta_matrix + x*SIZE_MU + y;
        
        // Invert nu-mu to r-theta
        float alpha = (256.0f * mu * mu) / (27 * (nu * nu * nu * nu));
        float beta = __powf((1 + __powf(1 + alpha, 0.5f)), (2.0f/3));
        float gamma = __powf(alpha, (1.0f/3));
        float zeta = __powf( ((beta*beta + beta*gamma + gamma*gamma) / beta), 
                             1.5f
                           ) / 2;
        (*r) = 4 * zeta / (nu * (1 + zeta) * (1 + __powf(2*zeta - 1, 0.5f)));
        (*sin_theta) = __powf((*r) * nu, 0.5f);
        
        // These may be needed for plotting. Decide what to do with them later.
        // float delta = __powf( (4 - 3*(*sin_theta)*(*sin_theta)), 0.5 );
        // float gridx = (*r) * __powf(1 - (*sin_theta)*(*sin_theta), 0.5);
        // float gridy = (*r) * sin_theta;
    }
}

// In the computation of the h-matrices, the existing matrices can be 
// overwritten. Beyond this point, only the h-matrices are actually used.
__global__ void
compute_h(float *h_nu_matrix, float *h_phi_matrix, float *h_mu_matrix, 
          float *r_matrix, float *sin_theta_matrix)
{
    // Get thread position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    // Check limits on x and y
    if(x >= SIZE_NU || y >= SIZE_MU) return;
    
    float *h_nu = h_nu_matrix + x*SIZE_MU + y;
    float *h_phi = h_phi_matrix + x*SIZE_MU + y;
    float *h_mu = h_mu_matrix + x*SIZE_MU + y;
    float r = r_matrix[x*SIZE_MU + y];
    float sin_theta = sin_theta_matrix[x*SIZE_MU + y];
    
    float delta = __powf( (4 - 3*sin_theta*sin_theta), 0.5 );
    
    // Assume RI is #defined somewhere.
    (*h_nu) = (r * r) / (RI * sin_theta * delta);
    (*h_phi) = r * sin_theta;
    (*h_mu) = (r * r * r) / (RI * RI * delta);
}
// Note: 64KB of const memory => 64K/3 ~ 21K => 5461 elements per h-matrix
//                            => matrix side length = 73 => abysmal
// That is, 64KB allows only a maximum simulation size of 73x73.
// And this is excluding epsilon and mu matrices which will also come into the 
// mix later on.

// Update for H_nu
__global__ void
update_H_nu(struct Data *d) {
    // Get thread position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    // Check limits on x and y
    if(x >= SIZE_NU || y >= SIZE_MU-1) return;
    
    // Compute coefficients
    int index = x*(SIZE_MU-1) + y;
    float cH = 1;
    float h_phi_prev = d->h_phi[index];
    float h_phi_next = d->h_phi[index + 1];
    float h_phi_avg = (h_phi_prev + h_phi_next) / 2;
    float h_mu_avg = (d->h_mu[index] + d->h_mu[index + 1]) / 2;
    // Sc_mu and imp0 should be __device__ const float somewhere
    // MU_R is #defined somewhere
    float cE = Sc_mu / (imp0 * MU_R * h_phi_avg * h_mu_avg);
    
    // Now for the actual update
    float *H_nu = d->H_nu + index;
    float E_phi_prev = d->E_phi[index];
    float E_phi_next = d->E_phi[index + 1];
    (*H_nu) = cH * (*H_nu) + cE * (  h_phi_next * E_phi_next
                                   - h_phi_prev * E_phi_prev );
}

// Update for H_phi
__global__ void
update_H_phi(struct Data *d) {
    // Get thread position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    // Check limits on x and y
    if(x >= SIZE_NU-1 || y >= SIZE_MU-1) return;
    
    // Compute coefficients
    // Note: column width of H_phi is SIZE_MU-1, but column width of h-matrices
    // are all SIZE_MU.
    int index = x*(SIZE_MU-1) + y;
    float cH = 1;
    float h_nu = (  d->h_nu[index] + d->h_nu[index + SIZE_MU + 1]
                  + d->h_nu[index + 1] + d->h_nu[index + SIZE_MU] ) / 4;
    float h_mu = (  d->h_mu[index] + d->h_mu[index + SIZE_MU + 1]
                  + d->h_mu[index + 1] + d->h_mu[index + SIZE_MU] ) / 4;
    float h_nu_fwd_avg = (  d->h_nu[index + SIZE_MU + 1]
                          + d->h_nu[index + 1]           ) / 2;
    float h_nu_bwd_avg = (d->h_nu[index] + d->h_nu[index + SIZE_MU]) / 2;
    float cE = - Sc_mu / (imp0 * MU_R * h_nu * h_mu);

    // Final update equation
    float *H_phi = d->H_phi + index;
    float E_nu_fwd = d->E_nu[index + 1];
    float E_nu_bwd = d->E_nu[index];
    (*H_phi) = cH * (*H_phi) + cE * (  h_nu_fwd_avg * E_nu_fwd
                                     - h_nu_bwd_avg * E_nu_bwd );
}

// Update for H_mu
__global__ void
update_H_mu(struct Data *d) {
    // Get thread position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    // Check limits on x and y
    if(x >= SIZE_NU-1 || y >= SIZE_MU) return;
    
    // Compute coefficients
    int index = x*SIZE_MU + y;
    float cH = 1;
    float h_nu_avg = (d->h_nu[index] + d->h_nu[index + SIZE_MU]) / 2;
    float h_phi_prev = d->h_phi[index];
    float h_phi_next = d->h_phi[index + SIZE_MU];
    float h_phi_avg = (h_phi_prev + h_phi_next) / 2;
    // Sc_nu also has to be __device__ const float somewhere.
    float cE = - Sc_nu / (imp0 * MU_R * h_nu_avg * h_phi_avg);
    
    // Final update equation
    float *H_mu = d->H_mu + index;
    float E_phi_prev = d->E_phi[index];
    float E_phi_next = d->E_phi[index + SIZE_MU];
    (*H_mu) = cH * (*H_mu) + cE * (  h_phi_next * E_phi_next
                                   - h_phi_prev * E_phi_prev );
}

// Update for E_nu
__global__ void
update_E_nu(struct Data *d) {
    // Get thread position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    // Check limits on x and y
    if(x >= SIZE_NU-1 || y == 0 || y >= SIZE_MU-1) return;
    
    // Compute coefficients
    int index = x*SIZE_MU + y;
    float cE = 1;
    float h_phi_avg = (d->h_phi[index] + d->h_phi[index + SIZE_MU]) / 2;
    float h_mu_avg = (d->h_mu[index] + d->h_mu[index + SIZE_MU]) / 2;
    float h_phi_fwd_avg = (  d->h_phi[index]
                           + d->h_phi[index + SIZE_MU + 1]
                           + d->h_phi[index + SIZE_MU]
                           + d->h_phi[index + 1]           ) / 4;
    float h_phi_bwd_avg = (  d->h_phi[index]
                           + d->h_phi[index + SIZE_MU - 1]
                           + d->h_phi[index + SIZE_MU]
                           + d->h_phi[index - 1]           ) / 4;
    // EPSILON_R to be #defined somewhere
    float cH = - Sc_mu * imp0 / (EPSILON_R * h_phi_avg * h_mu_avg);
    
    // Final update equation
    float *E_nu = d->E_nu + index;
    float H_phi_next = d->H_phi[index];
    float H_phi_prev = d->H_phi[index - 1];
    (*E_nu) = cE * (*E_nu) + cH * (  h_phi_fwd_avg * H_phi_next
                                   - h_phi_bwd_avg * H_phi_prev );
}

// Update for E_phi
__global__ void
update_E_phi(struct Data *d, int t) {
    // Get thread position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    // Check limits on x and y
    if(x == 0 || x >= SIZE_NU-1 || y == 0 || y >= SIZE_MU-1) return;
    
    // Compute coefficients
    int index = x*SIZE_MU + y;
    float cE = 1;
    float h_nu = d->h_nu[index];
    float h_mu = d->h_mu[index];
    float h_nu_fwd_avg = (d->h_nu[index + 1] + h_nu) / 2;
    float h_nu_bwd_avg = (h_nu + d->h_nu[index - 1]) / 2;
    float h_mu_fwd_avg = (d->h_mu[index + SIZE_MU] + h_mu) / 2;
    float h_mu_bwd_avg = (h_mu + d->h_mu[index - SIZE_MU]) / 2;
    float cH_nu = Sc_mu * imp0 / (EPSILON_R * h_nu * h_mu);
    float cH_mu = - Sc_nu * imp0 / (EPSILON_R * h_nu * h_mu);
    
    // Final update equation
    float *E_phi = d->E_phi + index;
    float H_nu_next = d->H_nu[index];
    float H_nu_prev = d->H_nu[index - 1];
    float H_mu_next = d->H_mu[index];
    float H_mu_prev = d->H_mu[index - SIZE_MU];
    (*E_phi) =   cE * (*E_phi)
               + cH_nu * (h_nu_fwd_avg * H_nu_next - h_nu_bwd_avg * H_nu_prev)
               + cH_mu * (h_mu_fwd_avg * H_mu_next - h_mu_bwd_avg * H_mu_prev);

    // Needs definition of SOURCE_LOCATION
    E_phi[SOURCE_LOCATION] = __expf(-(t-30) * (t-30) / 100.0);
}

int main(int argc, char **argv)
{
    struct Data data;
    
    std::cout<<"Program started\n";
    
    // TODO: Error handling!
    // Declare r, theta matrices
    hipMalloc((void **)&data.r, SIZE_NU * SIZE_MU * sizeof(float));
    hipMalloc((void **)&data.sin_theta, SIZE_NU * SIZE_MU * sizeof(float));
    
    std::cout<<"Allocated memory for r and theta matrices\n";
    
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    
    // Launch kernel to compute r and theta
    dim3 threads(16, 16);
    dim3 blocks(SIZE_NU / 16, SIZE_MU / 16);
    compute_r_theta<<<blocks, threads, 0, stream1>>>(data.r, data.sin_theta);
    
    std::cout<<"Done computing r and theta\n";
    
    // Allocate memory for h-matrices
    hipMalloc((void **)&data.h_nu, SIZE_NU * SIZE_MU * sizeof(float));
    hipMalloc((void **)&data.h_phi, SIZE_NU * SIZE_MU * sizeof(float));
    hipMalloc((void **)&data.h_mu, SIZE_NU * SIZE_MU * sizeof(float));
    
    std::cout<<"Allocated memory for h-matrices\n";
    
    // Launch kernel to compute h-matrices
    compute_h<<<blocks, threads, 0, stream1>>>(data.h_nu, data.h_phi, data.h_mu,
                                               data.r, data.sin_theta);
    
    std::cout<<"Done computing h matrices\n";

    // Declare host versions of E and H matrices
    // Might want to use HostAlloc later on so that copying to this can be made
    // asynchronous
    float E_nu[SIZE_NU-1][SIZE_MU] = { 0.0f };
    float E_phi[SIZE_NU][SIZE_MU] = { 0.0f };
    // E_mu is zero
    float H_nu[SIZE_NU][SIZE_MU-1] = { 0.0f };
    float H_phi[SIZE_NU-1][SIZE_MU-1] = { 0.0f };
    float H_mu[SIZE_NU-1][SIZE_MU] = { 0.0f };
    
    std::cout<<"Created host field matrices\n";
     
    // Allocate memory on the device for field matrices
    hipMalloc((void **)&data.E_nu, (SIZE_NU-1) * SIZE_MU * sizeof(float));
    hipMalloc((void **)&data.E_phi, SIZE_NU * SIZE_MU * sizeof(float));
    hipMalloc((void **)&data.H_nu, SIZE_NU * (SIZE_MU-1) * sizeof(float));
    hipMalloc((void **)&data.H_phi, (SIZE_NU-1) * (SIZE_MU-1) * sizeof(float));
    hipMalloc((void **)&data.H_mu, (SIZE_NU-1) * SIZE_MU * sizeof(float));
    
    std::cout<<"Allocated memory for device field matrices\n";
    
    // Copy host matrices to device in order to set zero
    hipMemcpy(data.E_nu, E_nu, (SIZE_NU-1) * SIZE_MU * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(data.E_phi, E_phi, SIZE_NU * SIZE_MU * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(data.H_nu, H_nu, SIZE_NU-1 * (SIZE_MU-1) * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(data.H_phi, H_phi, (SIZE_NU-1) * (SIZE_MU-1) * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(data.H_mu, H_mu, SIZE_NU * (SIZE_MU-1) * sizeof(float),
               hipMemcpyHostToDevice);
    
    std::cout<<"Done setting zero to device field matrices\n";
    
    // Copy pointers to device
    // One-time minor expense for increased ease of access...
    hipMalloc((void **)&data.d, sizeof(struct Data));
    hipMemcpy(data.d, &data, sizeof(data), hipMemcpyHostToDevice);
    
    // Start stepping to update E and H
    /* Note: Maybe we should compute the h-matrices also every single time. In
       all probability, considering that memory access can take upto 500 clock
       cycles, computing h-values each time might be faster - only benchmarking
       will tell */
    // Each update equation occurs as a separate kernel
    int t = 0;
    hipStream_t stream2, stream3;
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    
    std::cout<<"Starting time stepping...\n";
    
    while(t < MAXTIME) {
        // Launch update kernels
        // Note that H_nu, H_mu and E_phi form one set of independent equations
        // while H_phi and E_nu form another set. Therefore, they can be run in
        // parallel. Moreover, the update of H_nu and H_mu can also be run in
        // parallel.
        update_H_nu<<<blocks, threads, 0, stream2>>>(data.d);
        update_H_phi<<<blocks, threads, 0, stream1>>>(data.d);
        update_H_mu<<<blocks, threads, 0, stream3>>>(data.d);
        update_E_nu<<<blocks, threads, 0, stream1>>>(data.d);
        // E_phi should wait for both streams 2 and 3. We wait only for 3 since
        // it will be forced to wait for 2 if it is to run on 2.
        hipStreamSynchronize(stream3);
        update_E_phi<<<blocks, threads, 0, stream2>>>(data.d, t);
        
        // Make host wait until all updates are complete
        hipDeviceSynchronize();
        
        // This promises to be deadly slow. We'll need to figure out some
        // intelligent way of running this asynchronously. Lower frequency
        // requirement will help a lot. Benchmarking will tell us what the
        // highest possible frequency with zero lag will be.
        
        if(t % 5 == 0) {
            // Fetch from GPU
            hipMemcpy(E_phi, data.E_phi, SIZE_NU * SIZE_MU * sizeof(float), 
                       hipMemcpyDeviceToHost);
            
            // Write into file
            std::fstream f;
            char filename[50];
            sprintf(filename, "output/dipole/time-step-%d.txt", t);
            f.open(filename, std::fstream::out);
            f<<SIZE_NU<<std::endl<<SIZE_MU<<std::endl;
            for(int i=0 ; i < SIZE_NU ; i++) {
                for(int j=0 ; j < SIZE_MU ; j++ ) {
                    f<<E_phi[i][j]<<std::endl;
                }
            }
            f.close();
        }
        
        // Increment time step
        t++;
    }
    
    std::cout<<"All done\n";
    
    // Clean up
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipDeviceReset();
    return 0;
}

